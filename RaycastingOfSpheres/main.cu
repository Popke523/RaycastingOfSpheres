#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include "imgui.h"
#include "imgui_impl_glfw.h"
#include "imgui_impl_opengl3.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include ""
#include "kernel.h"

#include <iostream>

#include "random_objects_generator.h"

void framebuffer_size_callback(GLFWwindow *window, int width, int height);
void processInput(GLFWwindow *window);

// settings
const unsigned int SCR_WIDTH = 1280;
const unsigned int SCR_HEIGHT = 720;

const int N_SPHERES = 200;
const int N_LIGHTS = 20;

const char *vertexShaderSource = R"(
#version 330 core
layout (location = 0) in vec3 aPos;
layout (location = 1) in vec3 aColor;
layout (location = 2) in vec2 aTexCoord;

out vec3 ourColor;
out vec2 TexCoord;

void main()
{
	gl_Position = vec4(aPos, 1.0);
	ourColor = aColor;
	TexCoord = vec2(aTexCoord.x, aTexCoord.y);
}
)";

const char *fragmentShaderSource = R"(
#version 330 core
out vec4 FragColor;

in vec3 ourColor;
in vec2 TexCoord;

// texture sampler
uniform sampler2D texture1;

void main()
{
	FragColor = texture(texture1, TexCoord);
}
)";

// Function to compile shaders
GLuint compileShader(GLenum type, const char *source)
{
    GLuint shader = glCreateShader(type);
    glShaderSource(shader, 1, &source, NULL);
    glCompileShader(shader);

    int success;
    char infoLog[512];
    glGetShaderiv(shader, GL_COMPILE_STATUS, &success);
    if (!success)
    {
        glGetShaderInfoLog(shader, 512, NULL, infoLog);
        std::cerr << "Shader compilation failed: " << infoLog << std::endl;
    }
    return shader;
}

int main()
{
    hipSetDevice(0);

    // glfw: initialize and configure
    // ------------------------------
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

#ifdef __APPLE__
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

    // glfw window creation
    // --------------------
    GLFWwindow *window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);
    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

    // glad: load all OpenGL function pointers
    // ---------------------------------------
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }

    // Initialize ImGui
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO &io = ImGui::GetIO();
    (void)io;

    // Set up ImGui backends
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 330");

    // Set up ImGui style
    ImGui::StyleColorsDark();

    // build and compile our shader zprogram
    // ------------------------------------
    GLuint vertexShader = compileShader(GL_VERTEX_SHADER, vertexShaderSource);
    GLuint fragmentShader = compileShader(GL_FRAGMENT_SHADER, fragmentShaderSource);
    GLuint shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);
    glDeleteShader(vertexShader);
    glDeleteShader(fragmentShader);

    float vertices[] = {
        // positions          // colors           // texture coords
         1.0f,  1.0f, 0.0f,   1.0f, 0.0f, 0.0f,   1.0f, 1.0f, // top right
         1.0f, -1.0f, 0.0f,   0.0f, 1.0f, 0.0f,   1.0f, 0.0f, // bottom right
        -1.0f, -1.0f, 0.0f,   0.0f, 0.0f, 1.0f,   0.0f, 0.0f, // bottom left
        -1.0f,  1.0f, 0.0f,   1.0f, 1.0f, 0.0f,   0.0f, 1.0f  // top left 
    };
    unsigned int indices[] = {
        0, 1, 3, // first triangle
        1, 2, 3  // second triangle
    };
    unsigned int VBO, VAO, EBO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);

    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

    // position attribute
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void *)0);
    glEnableVertexAttribArray(0);
    // color attribute
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void *)(3 * sizeof(float)));
    glEnableVertexAttribArray(1);
    // texture coord attribute
    glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void *)(6 * sizeof(float)));
    glEnableVertexAttribArray(2);

    int width, height, nrChannels;

    width = SCR_WIDTH;
    height = SCR_HEIGHT;

    // load and create a texture 
    // -------------------------
    unsigned int texture;
    glGenTextures(1, &texture);
    glBindTexture(GL_TEXTURE_2D, texture); // all upcoming GL_TEXTURE_2D operations now have effect on this texture object
    // set the texture wrapping parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);	// set texture wrapping to GL_REPEAT (default wrapping method)
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    // set texture filtering parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glPixelStorei(GL_UNPACK_ROW_LENGTH, 0);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);

    //glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, texture, 0);


    //unsigned int rbo;
    //glGenRenderbuffers(1, &rbo);
    //glBindRenderbuffer(GL_RENDERBUFFER, rbo);
    //glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH24_STENCIL8, width, height);
    //glBindRenderbuffer(GL_RENDERBUFFER, 0);
    //glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_DEPTH_STENCIL_ATTACHMENT, GL_RENDERBUFFER, rbo);


    //GLuint fboId = 0;
    //glGenFramebuffers(1, &fboId);
    //glBindFramebuffer(GL_READ_FRAMEBUFFER, fboId);
    //glFramebufferTexture2D(GL_READ_FRAMEBUFFER, GL_COLOR_ATTACHMENT0,
    //    GL_TEXTURE_2D, texture, 0);

    hipGraphicsResource *m_TextureResource;

    hipGraphicsGLRegisterImage(&m_TextureResource, texture, GL_TEXTURE_2D,
        hipGraphicsRegisterFlagsSurfaceLoadStore);

    hipGraphicsMapResources(1, &m_TextureResource);

    hipArray *array;
    hipGraphicsSubResourceGetMappedArray(&array, m_TextureResource, 0, 0);

    hipResourceDesc desc;
    memset(&desc, 0, sizeof(hipResourceDesc));
    desc.resType = hipResourceTypeArray;
    desc.res.array.array = array;

    hipSurfaceObject_t surface;

    hipCreateSurfaceObject(&surface, &desc);

    srand(time(nullptr));

    camera camera;
    camera.position = make_float3(0.0f, 0.0f, 10.0f);
    camera.fov_degrees = 90.0f;
    camera.pitch_degrees = 0.0f;
    camera.yaw_degrees = 0.0f;

    
    sphere spheres[N_SPHERES];
    lightSource lightSources[N_LIGHTS];
    for (int i = 0; i < N_SPHERES; i++)
    {
        spheres[i] = random_sphere(-5.0f, 5.0f, 0.1f, 0.15f);	
    }
    for (int i = 0; i < N_LIGHTS; i++)
    {
        lightSources[i] = random_light_source(-10.0f, 10.0f);
    }

	sphere *deviceSpheres;
	hipMalloc(&deviceSpheres, sizeof(spheres));
	hipMemcpy(deviceSpheres, &spheres, sizeof(spheres), hipMemcpyHostToDevice);

    lightSource *deviceLightSources;
    hipMalloc(&deviceLightSources, sizeof(lightSources));
	hipMemcpy(deviceLightSources, &lightSources, sizeof(lightSources), hipMemcpyHostToDevice);


    glBindFramebuffer(GL_FRAMEBUFFER, 0);

    // render loop
    // -----------
    while (!glfwWindowShouldClose(window))
    {
        // input
        // -----
        processInput(window);

        renderTestKernelLauncher(surface, width, height, camera, deviceSpheres, N_SPHERES, deviceLightSources, N_LIGHTS);
        //hipGraphicsUnmapResources(1, &m_TextureResource);

        // render
        // ------
        glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        glBindTexture(GL_TEXTURE_2D, texture);

        glUseProgram(shaderProgram);
		glBindVertexArray(VAO);
		glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);


        // Start new ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        // Create GUI
        ImGui::SetNextWindowPos(ImVec2(16.0f, 16.0f), ImGuiCond_Once);
		ImGui::SetNextWindowSize(ImVec2(320.0f, 200.0f), ImGuiCond_Once);
        ImGui::Begin("Simple GUI");

        ImGui::Text("Frame rate: %.1f FPS", ImGui::GetIO().Framerate);

		// imgui sliders for camera settings
		ImGui::SliderFloat("Camera X", &camera.position.x, -10.0f, 10.0f);
		ImGui::SliderFloat("Camera Y", &camera.position.y, -10.0f, 10.0f);
		ImGui::SliderFloat("Camera Z", &camera.position.z, -10.0f, 10.0f);
		ImGui::SliderFloat("Camera Pitch", &camera.pitch_degrees, -90.0f, 90.0f);
		ImGui::SliderFloat("Camera Yaw", &camera.yaw_degrees, -180.0f, 180.0f);
        ImGui::SliderFloat("Camera FOV", &camera.fov_degrees, 30.0f, 150.0f);

        ImGui::SetCursorPosX(0.0f);
        ImGui::SetCursorPosY(0.0f);

        ImGui::End();

        // Render ImGui
        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());


        // bind Texture
        // glBindTexture(GL_TEXTURE_2D, texture);

        // render container
        //glBindVertexArray(VAO);
        //glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);

        // glfw: swap buffers and poll IO events (keys pressed/released, mouse moved etc.)
        // -------------------------------------------------------------------------------
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    //// optional: de-allocate all resources once they've outlived their purpose:
    //// ------------------------------------------------------------------------
    //glDeleteVertexArrays(1, &VAO);
    //glDeleteBuffers(1, &VBO);
    //glDeleteBuffers(1, &EBO);

    // glfw: terminate, clearing all previously allocated GLFW resources.
    // ------------------------------------------------------------------
    glfwTerminate();
    return 0;
}

// process all input: query GLFW whether relevant keys are pressed/released this frame and react accordingly
// ---------------------------------------------------------------------------------------------------------
void processInput(GLFWwindow *window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
}

// glfw: whenever the window size changed (by OS or user resize) this callback function executes
// ---------------------------------------------------------------------------------------------
void framebuffer_size_callback(GLFWwindow *window, int width, int height)
{
    // make sure the viewport matches the new window dimensions; note that width and 
    // height will be significantly larger than specified on retina displays.
    glViewport(0, 0, width, height);
}